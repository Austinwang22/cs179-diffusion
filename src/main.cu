#include "hip/hip_runtime.h"
#include "diffusion/DiffusionLoader.cu"

#include "ErrorCheck.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

void test_forward_with_weights() {
    using DC = DiffusionConfig;
    const int B = 1;                      // batch size
    const int H = DC::img_resolution;
    const int W = DC::img_resolution;
    const int C = DC::in_channels;       // typically 1
    const int numel = B * C * H * W;
    const size_t bytes = numel * sizeof(__hip_bfloat16);

    // ------------------------------
    // Load weights from safetensors
    // ------------------------------
    DiffusionLoader loader;

    const std::string weight_path = loader.get_model_dir();
    // DiffusionWeights weights = loader.load_diffusion_weights(weight_path);

    // ------------------------------
    // Initialize model and load weights
    // ------------------------------
    // UNetBF16 model(DC::img_resolution, DC::t_emb_dim);
    // model.load_weights(weights);
    std::shared_ptr<UNetBF16> model = loader.load_diffusion_weights(weight_path);

    // ------------------------------
    // Allocate and fill dummy input
    // ------------------------------
    CudaBuffer input(bytes);
    std::vector<__hip_bfloat16> h_input(numel, __float2bfloat16(1.0f));  // Constant input
    checkCuda(hipMemcpy(input.data, h_input.data(), bytes, hipMemcpyHostToDevice));

    // ------------------------------
    // Allocate and fill dummy time embedding
    // ------------------------------
    int32_t h_tstamp[B] = {32};  // arbitrary timestep
    int32_t *d_tstamp;
    checkCuda(hipMalloc(&d_tstamp, sizeof(int32_t) * B));
    checkCuda(hipMemcpy(d_tstamp, h_tstamp, sizeof(int32_t) * B, hipMemcpyHostToDevice));

    // ------------------------------
    // Perform forward pass
    // ------------------------------
    hipStream_t stream;
    checkCuda(hipStreamCreate(&stream));
    model->forward(static_cast<__hip_bfloat16 *>(input.data), h_tstamp, B, stream);
    checkCuda(hipStreamSynchronize(stream));

    // ------------------------------
    // Dump part of output
    // ------------------------------
    std::vector<__hip_bfloat16> h_out(numel);
    checkCuda(hipMemcpy(h_out.data(), input.data, bytes, hipMemcpyDeviceToHost));

    std::cout << "[Forward Output]: ";
    for (int i = 0; i < std::min(16, numel); ++i) {
        std::cout << __bfloat162float(h_out[i]) << " ";
    }
    std::cout << std::endl;

    // ------------------------------
    // Cleanup
    // ------------------------------
    checkCuda(hipFree(d_tstamp));
    checkCuda(hipStreamDestroy(stream));
}

int main() {
    test_forward_with_weights();
    return 0;
}

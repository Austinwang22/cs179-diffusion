#include "hip/hip_runtime.h"
#include "diffusion/DiffusionLoader.h"
#include "diffusion/DiffusionConfig.h"

#include "diffusion/DiffusionUNet.cuh"

#include "ErrorCheck.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

void test_forward_with_weights() {
    using DC = DiffusionConfig;
    const int B = 1;                      // batch size
    const int H = DC::img_resolution;
    const int W = DC::img_resolution;
    const int C = DC::in_channels;       // typically 1
    const int numel = B * C * H * W;
    const size_t bytes = numel * sizeof(__hip_bfloat16);

    // ------------------------------
    // Load weights from safetensors
    // ------------------------------
    DiffusionLoader loader;

    const std::string weight_path = loader.get_model_dir();

    DiffusionWeights weights = loader.load_diffusion_weights(weight_path);

    // ------------------------------
    // Initialize model and load weights
    // ------------------------------
    UNetBF16 model(DC::img_resolution, DC::t_emb_dim);
    model.load_weights(weights);

    // ------------------------------
    // Allocate and fill dummy input
    // ------------------------------
    CudaBuffer input(bytes);
    std::vector<__hip_bfloat16> h_input(numel, __float2bfloat16(1.0f));  // Constant input
    checkCuda(hipMemcpy(input.data, h_input.data(), bytes, hipMemcpyHostToDevice));

    // ------------------------------
    // Allocate and fill dummy time embedding
    // ------------------------------
    int32_t h_tstamp[B] = {32};  // arbitrary timestep
    int32_t *d_tstamp;
    checkCuda(hipMalloc(&d_tstamp, sizeof(int32_t) * B));
    checkCuda(hipMemcpy(d_tstamp, h_tstamp, sizeof(int32_t) * B, hipMemcpyHostToDevice));

    // ------------------------------
    // Perform forward pass
    // ------------------------------

    hipStream_t stream;
    checkCuda(hipStreamCreate(&stream));
    model.forward(static_cast<__hip_bfloat16 *>(input.data), h_tstamp, B, stream);
    checkCuda(hipStreamSynchronize(stream));

    // ------------------------------
    // Dump part of output
    // ------------------------------
    std::vector<__hip_bfloat16> h_out(numel);
    checkCuda(hipMemcpy(h_out.data(), input.data, bytes, hipMemcpyDeviceToHost));

    std::cout << "[Forward Output]: ";
    for (int i = 0; i < std::min(16, numel); ++i) {
        std::cout << __bfloat162float(h_out[i]) << " ";
    }
    std::cout << std::endl;

    // ------------------------------
    // Cleanup
    // ------------------------------
    checkCuda(hipFree(d_tstamp));
    checkCuda(hipStreamDestroy(stream));
}

int main() {
    test_forward_with_weights();
    return 0;
}


// ======== random weights forward pass ============================ ///
// // main.cpp – toy inference driver for the bf16 diffusion U-Net
// // ------------------------------------------------------------------
// // compile (CUDA 12+):
// // nvcc -std=c++17 -O3 -arch=sm_80 main.cpp -lcudnn -lcublas -lcurand
// // ------------------------------------------------------------------
// #include <hip/hip_runtime.h>
// #include <hiprand.h>
// #include <iostream>
// #include <vector>

// #include "diffusion/DiffusionConfig.h"
// #include "diffusion/DiffusionUNet.cuh"

// using namespace dm;

// // ------------------------------------------------------------------
// // quick helper: fill a raw bf16 buffer with U(0,1) noise
// // ------------------------------------------------------------------
// __global__ void rand2bf16(float *src, __hip_bfloat16 *dst, size_t n)
// {
//     size_t i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n) dst[i] = __float2bfloat16(src[i]);
// }

// void fill_random_bf16(void *dev_ptr, size_t elems, hipStream_t st)
// {
//     static hiprandGenerator_t gen = nullptr;
//     if (!gen) hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//     hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

//     CudaBuffer tmp(elems * sizeof(float));
//     hiprandGenerateUniform(gen, static_cast<float*>(tmp.data), elems);

//     dim3 blk(256);
//     dim3 grd((elems + blk.x - 1) / blk.x);
//     rand2bf16<<<grd, blk, 0, st>>>(static_cast<float*>(tmp.data),
//                                    static_cast<__hip_bfloat16*>(dev_ptr),
//                                    elems);
// }

// // ------------------------------------------------------------------
// // run a single forward pass through the network
// // ------------------------------------------------------------------
// void forward_once(UNetBF16 &net, int B, hipStream_t st)
// {
//     constexpr int H = DiffusionConfig::img_resolution;
//     constexpr int W = DiffusionConfig::img_resolution;

//     // x_noisy  (BF16)
//     CudaBuffer x_buf(size_t(B) * H * W * sizeof(__hip_bfloat16));
//     fill_random_bf16(x_buf.data, x_buf.size / sizeof(__hip_bfloat16), st);

//     // dummy timesteps (all zeros just for sanity-check)
//     std::vector<int32_t> t_host(B, 0);

//     net.forward(static_cast<__hip_bfloat16*>(x_buf.data), t_host.data(), B, st);
//     hipStreamSynchronize(st);
// }

// int main()
// {
//     hipStream_t st{}; hipStreamCreate(&st);

//     // 1. build the diffusion U-Net
//     UNetBF16 net(28, 128); //resolution, time embedding

//     // 2. forward pass (no weight randomisation necessary for a compile test)
//     forward_once(net, 1, st); // single batch

//     std::cout << "Inference completed (random weights = implicit zeros).\n";

//     hipStreamDestroy(st);
//     return 0;
// }
